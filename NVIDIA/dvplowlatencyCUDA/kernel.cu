#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"

// Utility macros
#define DIVUP(A,B) ( (A)%(B) == 0 ? (A)/(B) : ((A) / (B) + 1) )

// The thread block size
#define BLOCK_SIZE_W 32
#define BLOCK_SIZE_H 32

surface<void, 2> inSurfRef;
surface<void, 2> outSurfRef;
__global__ void Copy_kernel(unsigned int width, unsigned int height)
									  
{
	// Indices into the image data
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;  
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < width && y < height) {
		uchar4 data; 

		// Read from input surface texture
		surf2Dread(&data, inSurfRef, x * 4, y);

		// Write to outputsurface texture
		surf2Dwrite(data, outSurfRef, x * 4, y);
	}
}

extern "C" void CopyVideoInputToOuput(hipArray *pIn, hipArray *pOut, 
	                                  unsigned int width, unsigned int height)
{
	// Bind arrays to surface reference
	checkCudaErrors(cudaBindSurfaceToArray(inSurfRef, pIn));
	checkCudaErrors(cudaBindSurfaceToArray(outSurfRef, pOut));

	// Set the block size
    dim3 BlockSz(BLOCK_SIZE_W, BLOCK_SIZE_H, 1);

    // Set the grid size
    dim3 GridSz(DIVUP(width, BLOCK_SIZE_W), DIVUP(height, BLOCK_SIZE_H), 1);

	// Execute the kernel
    Copy_kernel<<<GridSz,BlockSz>>>(width, height);

    // Wait for kernel processing to complete for all threads.
    hipCtxSynchronize();
}

