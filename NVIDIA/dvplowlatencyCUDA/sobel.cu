#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"

texture <float, 2, hipReadModeElementType> inTex;
surface<void, 2> outSurfRef;

//Kernel for x direction sobel
__global__ void implement_x_sobel(int width, int height, int widthStep)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Make sure that thread is inside image bounds
    if(x<width && y<height)
    {
		uchar4 rgba;

		float output_value_x = (-1 * tex2D(inTex, x - 1, y - 1)) + (0 * tex2D(inTex, x, y - 1)) + (1 * tex2D(inTex, x + 1, y - 1))
			                 + (-2 * tex2D(inTex, x - 1, y))     + (0 * tex2D(inTex, x, y))     + (2 * tex2D(inTex, x + 1, y))
			                 + (-1 * tex2D(inTex, x - 1, y + 1)) + (0 * tex2D(inTex, x, y + 1)) + (1 * tex2D(inTex, x + 1, y + 1));

		float output_value_y = (-1 * tex2D(inTex, x - 1, y - 1)) + (-2 * tex2D(inTex, x, y - 1)) + (1 * tex2D(inTex, x + 1, y - 1))
			                 + (0 * tex2D(inTex, x - 1, y)) + (0 * tex2D(inTex, x, y)) + (0 * tex2D(inTex, x + 1, y))
			                 + (-1 * tex2D(inTex, x - 1, y + 1)) + (2 * tex2D(inTex, x, y + 1)) + (1 * tex2D(inTex, x + 1, y + 1));

		float output_value = sqrt((output_value_x * output_value_x) + (output_value_y * output_value_y));

		rgba.x = output_value * 255;
		rgba.y = output_value * 255;
		rgba.z = output_value * 255;
		rgba.w = output_value * 255;

		surf2Dwrite(rgba, outSurfRef, x * 4, y);
    }
}

extern "C" void DoSobel(hipArray *greyImage, hipArray *outImage, unsigned int width, unsigned int height, unsigned int widthStep)
{
	// Bind arrays to surface reference
	checkCudaErrors(hipBindTextureToArray(inTex, greyImage));
	checkCudaErrors(cudaBindSurfaceToArray(outSurfRef, outImage));
	
	const dim3 blocksize(16,16);
	const dim3 gridsize((width + blocksize.x - 1) / blocksize.x, 
		                (height + blocksize.y - 1) / blocksize.y, 
						1);

    implement_x_sobel<<<gridsize,blocksize>>>(width, height, widthStep/sizeof(float));

	hipDeviceSynchronize();
}
