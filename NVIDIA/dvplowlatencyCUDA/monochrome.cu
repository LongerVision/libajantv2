#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"

surface<void, 2> inSurfRef;
surface<void, 2> outSurfRef;

__global__ void rgba_to_greyscale(unsigned int width, unsigned int height)
{
	// Indices into the image data
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < width && y < height) {
		uchar4 rgba;

		// Read from input surface texture
		surf2Dread(&rgba, inSurfRef, x * 4, y);

		float channelSum = (.299f * rgba.x + .587f * rgba.y + .114f * rgba.z) / 255.0f;

		// Write to output surface texture
		surf2Dwrite(channelSum, outSurfRef, x * 4, y);
	}
}

extern "C" void ConvertRGBAToGreyscale(hipArray *rgbaImage,
                                       hipArray *greyImage,
                                       unsigned int width,
                                       unsigned int height)
{
	// Bind arrays to surface reference
	checkCudaErrors(cudaBindSurfaceToArray(inSurfRef, rgbaImage));
	checkCudaErrors(cudaBindSurfaceToArray(outSurfRef, greyImage));

	//You must fill in the correct sizes for the blockSize and gridSize
	//currently only one block with one thread is being launched
	const dim3 blockSize(24, 24, 1);
	const dim3 gridSize((width/16), (height/16) , 1);
	rgba_to_greyscale<<<gridSize, blockSize>>>(width, height);

	// Wait for kernel processing to complete for all threads.
	hipCtxSynchronize();;
}